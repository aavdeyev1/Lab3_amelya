#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.y;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx] = threadIdx.x;
}

__global__ void kernel3( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx] = idx;
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockDim.x*blockIdx.x + blockIdx.y;
    
}

__global__ void kernel5( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.y;
}

__global__ void kernel6( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.x;
}


