#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.y;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.x + blockIdx.y*blockDim.x;
}

__global__ void kernel3( int *a, int dimx, int dimy )
{ // DOne
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx] = idx;
}

__global__ void kernel4( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
    a[idx] = blockDim.x*blockIdx.y + blockIdx.x*ix;
    
}

__global__ void kernel5( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.y;
}

__global__ void kernel6( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < 16)
        a[idx] = blockIdx.x;
}


