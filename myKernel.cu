#include "hip/hip_runtime.h"
#include "myKernel.h"

__global__ void kernel( int *a, int dimx, int dimy )
{
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx)
        a[idx] = blockIdx.y;
}

// Please implement the following kernels2 through kernel6,
// in order to meet the requirements in the write-ups. 
__global__ void kernel2( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx)
        a[idx] = gridDim.x * blockIdx.y + blockIdx.x;
}

__global__ void kernel3( int *a, int dimx, int dimy )
{ // DOne
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;

    a[idx] = idx;
}

__global__ void kernel4( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx)
        a[idx] = threadIdx.x + blockDim.x*threadIdx.y;
    
}

__global__ void kernel5( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx)
        a[idx] = blockIdx.y;
}

__global__ void kernel6( int *a, int dimx, int dimy )
{ // Done
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*dimx + ix;
    if (ix < dimx)
        a[idx] = blockIdx.x;
}


